#include "hip/hip_runtime.h"
/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-3
 * Description: Activation Game 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include "graph.hpp"
 
#define INTMAX 1000000007

using namespace std;


ofstream outfile; // The handle for printing the output

/******************************Write your kerenels here ************************************/
/* In this kernel we are calculating last node of each level using the 
assumption that nodes are sequential on each level.*/
__global__ void findLastNodeOfEachLevel(int *d_lastNode,int firstNode, int lastNode, int *d_offset, int *csr){
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    node+=firstNode;
    if(node<=lastNode){
        /* Using offset and csr to compute the edges of each node in current level*/
        for(int i=d_offset[node]; i<d_offset[node+1];i++){
            atomicMax(&d_lastNode[0], csr[i]);
        }
    } 
}
       

/* Here we are activating level 0 nodes and calculating indegree of level 1. */
__global__ void activateLevel0(int firstNode, int lastNode, int V, int currLevel, int *aid,
 bool *activeness, int *csr, int *offset){

    int node = blockIdx.x * blockDim.x + threadIdx.x;
    node+=firstNode;
    if(node<=lastNode){
        aid[node]=0;
        activeness[node]=true;
        /* Using offset and csr to compute the edges of each node in current level*/
        for(int i=offset[node]; i<offset[node+1];i++){
            atomicInc((unsigned int *)&aid[csr[i]],INTMAX); //aid[csr[i]]++
        }
    }
}

/* Here we are activating level 1 - (L-1) nodes and calculating indegree of 
next level. */
__global__ void activateRestOfTheLevels(int firstNode, int lastNode, int i, int *offset, int *csr
, int *aid, int *apr, bool *activeness){

    int node = blockIdx.x * blockDim.x + threadIdx.x;
    node+=firstNode;
    if(node<=lastNode){
        // Applying rule 1. 
        if(aid[node]>= apr[node]){
            activeness[node]=true;   
        }
    }
}

/* Here we are applying second rule.*/
__global__ void deactivateRestOfTheLevels(int firstNode, int lastNode, int i, int *offset, int *csr,
int *aid, int *apr, bool *activeness){
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    node+=firstNode;
    if(node<=lastNode){
        // Here we are deactivating the nodes.
        if((node-1)>=firstNode && (node+1)<=lastNode && activeness[node-1]==false
            && activeness[node+1]==false){
            activeness[node]=false;
        }

        /* If node is still active then in that case increase active indegree
        of nodes of next level*/
        if(activeness[node]==true){
            for(int i=offset[node]; i<offset[node+1];i++){
                atomicInc((unsigned int *)&aid[csr[i]],INTMAX);//atomic_inc(); //aid[csr[i]]++
            }
        }
    }
}
    

/* In this kernel we are counting the active nodes on each level after 
doing all the processing. */
__global__ void countActiveNodes(int currLevel,int firstNode, int lastNode, 
  int *d_activeVertex, bool *activeness){

    int node = blockIdx.x * blockDim.x + threadIdx.x;
    node+=firstNode;

    if(node<=lastNode){
      if(activeness[node]==true){
        atomicInc((unsigned int *)&d_activeVertex[currLevel],INTMAX);
      }
    }

  }
    
/**************************************END*************************************************/



//Function to write result in output file
void printResult(int *arr, int V,  char* filename){
    outfile.open(filename);
    for(long int i = 0; i < V; i++){
        outfile<<arr[i]<<" ";   
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    int V ; // Number of vertices in the graph
    int E; // Number of edges in the graph
    int L; // number of levels in the graph

    //Reading input graph
    char *inputFilePath = argv[1];
    graph g(inputFilePath);

    //Parsing the graph to create csr list
    g.parseGraph();

    //Reading graph info 
    V = g.num_nodes();
    E = g.num_edges();
    L = g.get_level();


    //Variable for CSR format on host
    int *h_offset; // for csr offset
    int *h_csrList; // for csr
    int *h_apr; // active point requirement

    //reading csr
    h_offset = g.get_offset();
    h_csrList = g.get_csr();   
    h_apr = g.get_aprArray();
    
    // Variables for CSR on device
    int *d_offset;
    int *d_csrList;
    int *d_apr; //activation point requirement array
    int *d_aid; // acive in-degree array
    //Allocating memory on device 
    hipMalloc(&d_offset, (V+1)*sizeof(int));
    hipMalloc(&d_csrList, E*sizeof(int)); 
    hipMalloc(&d_apr, V*sizeof(int)); 
    hipMalloc(&d_aid, V*sizeof(int));

    //copy the csr offset, csrlist and apr array to device
    hipMemcpy(d_offset, h_offset, (V+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrList, h_csrList, E*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_apr, h_apr, V*sizeof(int), hipMemcpyHostToDevice);

    // variable for result, storing number of active vertices at each level, on host
    int *h_activeVertex;
    h_activeVertex = (int*)malloc(L*sizeof(int));
    // setting initially all to zero
    memset(h_activeVertex, 0, L*sizeof(int));

    // variable for result, storing number of active vertices at each level, on device
    int *d_activeVertex;
	hipMalloc(&d_activeVertex, L*sizeof(int));


/***Important***/

// Initialize d_aid array to zero for each vertex
// Make sure to use comments

/***END***/
double starttime = rtclock(); 

/*********************************CODE AREA*****************************************/

/* Init. d_activeVertex with 0*/
hipMemset(d_activeVertex, 0, (L)*sizeof(int));
bool *d_activeness;
/* activeness array tells us which nodes are active/unactive*/
hipMalloc(&d_activeness, (V)*sizeof(bool));
hipMemset(d_activeness, 0, (V)*sizeof(bool));

/* Find last node of level 0*/
int lastNodeOfLevel0=0;
while(h_apr[lastNodeOfLevel0]==0){
    lastNodeOfLevel0++;
}


/* Here we have declared all the variables that 
are required to create and build the array to store last node of each level.*/

int *h_lastNodeOfEachLevel; // stores last node of each level.

h_lastNodeOfEachLevel = (int *)malloc((L+1)*sizeof(int)); // L size
h_lastNodeOfEachLevel[L] = -1;

h_lastNodeOfEachLevel[0]=lastNodeOfLevel0-1;
int firstNode = 0;

int *h_lastNode, *d_lastNode;
hipMalloc(&d_lastNode, sizeof(int));
h_lastNode = (int *)malloc(sizeof(int));
h_lastNode[0] = lastNodeOfLevel0 -1;
hipMemcpy(d_lastNode, h_lastNode, sizeof(int), hipMemcpyHostToDevice);
int lastNode = lastNodeOfLevel0-1;

for(int i=0;i<L-1;i++){
    int totalThreads = h_lastNodeOfEachLevel[i] - firstNode + 1;
    int totalBlocks = (totalThreads + 1024-1)/1024;
    findLastNodeOfEachLevel<<<totalBlocks, 1024>>>(d_lastNode,firstNode, lastNode, d_offset, d_csrList);
    hipDeviceSynchronize();
    firstNode = lastNode + 1;
    hipMemcpy(h_lastNode, d_lastNode, sizeof(int), hipMemcpyDeviceToHost);
    h_lastNodeOfEachLevel[i+1] = h_lastNode[0];
    lastNode = h_lastNode[0];
    // h_lastNodeOfEachLevel[i+1]=lastNode;
}

/* now we have last nodes of each level stored in h_lastNodeOfEachLevel array*/


/* ---- process level 0 separately ------ */ 
int totalThreads = h_lastNodeOfEachLevel[0] - 0 + 1;
int totalBlocks = (totalThreads+1024-1)/1024;
firstNode = 0;
lastNode = h_lastNodeOfEachLevel[0];
activateLevel0<<<totalBlocks,1024>>>(firstNode,lastNode,V, 0, d_aid, d_activeness
, d_csrList, d_offset);
hipDeviceSynchronize();



/* ---- process level 1 to (L-1) now  ------  */
firstNode = h_lastNodeOfEachLevel[0]+1;
lastNode = h_lastNodeOfEachLevel[1];

for(int i=1; i<L;i++){
    totalThreads = lastNode - firstNode + 1;
    totalBlocks = (totalThreads+1024-1)/1024;
    // call activate 
    activateRestOfTheLevels<<<totalBlocks, 1024>>>(firstNode, lastNode, i, 
    d_offset, d_csrList, d_aid, d_apr, d_activeness);
    hipDeviceSynchronize();
    // call deactivate 
    deactivateRestOfTheLevels<<<totalBlocks, 1024>>>(firstNode, lastNode, i, 
    d_offset, d_csrList, d_aid, d_apr, d_activeness);
    hipDeviceSynchronize();
    firstNode = lastNode+1;
    lastNode = h_lastNodeOfEachLevel[i+1];
}
    


/* Calculate number of active nodes on each level and store it in d_activeVertex*/
int count = 0;
firstNode = 0;
lastNode = h_lastNodeOfEachLevel[0];

for(int i =0; i<L ; i++){
  totalThreads = lastNode - firstNode + 1;
  totalBlocks = (totalThreads + 1024 - 1)/1024;
  countActiveNodes<<<totalBlocks, 1024>>>(i , firstNode, lastNode, 
  d_activeVertex, d_activeness);
  hipDeviceSynchronize();
  firstNode = lastNode +1;
  lastNode = h_lastNodeOfEachLevel[i+1];
}
    
hipMemcpy(h_activeVertex, d_activeVertex, L*sizeof(int), hipMemcpyDeviceToHost);
   

     

/********************************END OF CODE AREA**********************************/
double endtime = rtclock();  
printtime("GPU Kernel time: ", starttime, endtime);  

// --> Copy C from Device to Host
char outFIle[30] = "./output.txt" ;
printResult(h_activeVertex, L, outFIle);
if(argc>2)
{
    for(int i=0; i<L; i++)
    {
        printf("level = %d , active nodes = %d\n",i,h_activeVertex[i]);
    }
}

    return 0;
}
